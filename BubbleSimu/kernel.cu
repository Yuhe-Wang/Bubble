#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <stdio.h>
//head files of C++ feature
#include <map>
#include <string>

#include <windows.h> //this is related to windows platform, used to launch another program

#define PI 3.141593
#define K_SPHERE (4.0/3.0*PI)
#define grid_size (N_bubble-1)/Block_size+1 //calculate how many blocks need to be launched
#define NULLD N_max_bubble_d
#define VEC_D thrust::raw_pointer_cast(&vec_bubbles[0])
//define the data type
class Bubble
{
public:
	float x,y,z;//coordinates of the center
	float R; //radius of the bubble
	float R3; //volume, used to reduce the error
	size_t ip; //index of pointing to
	size_t iped; //index of being pointed
};
typedef Bubble* PBubble;

// user-defined comparison operator that acts like less<int>,
struct compare_height
{
	__device__
		bool operator()(PBubble x, PBubble y)
	{
		return x->z > y->z; //let the bigger ones stand in front
	}
};

//global variables in the GPU, whose initial values are obtained from CPU
__constant__ float Hm_max;//horizontal move very step, unit mm
__constant__ float K_velocity; //the coefficient in the formula u=(2*g*rho/9/eta)*R^2
__constant__ float K_escape; //determine the critical condition for the bubble to escape
__constant__ float Width_container_d; //_d fix means it's a copy from the CPU, not unique in GPU
__constant__ float Height_container_d;
__constant__ size_t N_max_bubble_d;
__constant__ long *rands; //array of random numbers
__device__ int d_has_overlap=0; //overlap indicator
__device__ int d_multi_tree=0;
extern __shared__ Bubble bs[];

//default global constant value in CPU
size_t N_max_bubble=10000; //max number of bubbles
size_t N_init=0;  //how many bubbles in the container at beginning
size_t N_max_step=1000; //steps of evolution
size_t N_start_step= 1; //the step that begins to output data
int N_steps_output=10; //skip how many steps for one output
float N_inject_rate=9; //how many bubbles injected per dt, which may be a float number or even zero
float InitR=1.0; //initial radius of the injected bubble, unit mm, must larger than R_min
float Width_container=40; //unit mm
float Height_container=100; //unit mm
float K_interval=0; //the ratio of interval between injected bubbles to InitR, which reflects the density
float Init_height=100; //the ratio of height that is filled with bubbles at initial time.
int Block_size=256;
int Rand_seed=0;
int Init_var=0; //0 means to set distribution for R while 1 means for R
int Output_var=0; //0 means to output R while 1 means V=4/3*PI*R^3
int InitDistFolder=1; //chose a folder which contains the distribution dll file.

//global variables, change frequently
size_t N_bubble=0; // current number of bubbles
size_t N_reusable=0;
long In=0;

/***************************************Functions in the GPU******************************************************/

__device__ float random(long seed, long* ptr) //random number generator
{

	long In=16807*(seed%127773)-2836*(seed/127773);
	if(In<0) In+=2147483647;
	*ptr=In;
	return (float)In/2147483647;
}

__global__ void moveBubbles(PBubble* ptr_bubbles, size_t N_bubble)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x; //thread index
	if(i < N_bubble) //the thread index must be less than number of numbers
	{
		PBubble pb=ptr_bubbles[i];
		if(Hm_max>0) //move the bubbles in x-y plane
		{
			float x,y,theta,r;
			long seed=rands[i];//initialize the seed of random generator by the thread index
			do
			{
				theta=random(seed,&seed)*2*PI; // assume the horizontal moving vector is uniformly spread in a pie
				r=Hm_max;
				x=pb->x + r*cosf(theta);
				y=pb->y + r*sinf(theta);
			}while(x<0 || x> Width_container_d || y<0 || y> Width_container_d);
			pb->x = x;
			pb->y = y;
			rands[i]=seed;//store the seed back
		}	
		if(pb->z) pb->z +=K_velocity*pb->R*pb->R; //move up the bubble in z direction by K_velocity*R^2; 
		// set the invalid bubble's height to zero, so that it wouldn't affect the overlap detection
		if(pb->z > Height_container_d+ K_escape*pb->R) 
		{
			pb->z=0;
			d_has_overlap=1; //give a warning that there's bubble getting out of water
		}	
	}
}

__global__ void checkOverlap(PBubble* ptr_bubbles, size_t N_bubble)
{

	long base=blockIdx.x * blockDim.x;
	long i =base + threadIdx.x; //thread index
	bool first_overlap=false;

	if(i < N_bubble) //the thread index must be less than number of bubbles
	{	
		//load bubbles to shared memory
		PBubble sp=ptr_bubbles[i];
		bs[threadIdx.x].R=sp->R;
		bs[threadIdx.x].x=sp->x;
		bs[threadIdx.x].y=sp->y;
		bs[threadIdx.x].z=sp->z;
		__syncthreads(); //wait until all block threads finished loading

		float sR=bs[threadIdx.x].R;
		float sx=bs[threadIdx.x].x;
		float sy=bs[threadIdx.x].y;
		float sz=bs[threadIdx.x].z;
		float up_limit=sz+2*sR;
		float low_limit=sz-2*sR;

		////////////
		PBubble np;
		float nR,nx,ny,nz;
		//iterate to the left, which means larger z
		long j=i-1;
		while(j>=0)
		{

			if(j-base>=0 && j-base< blockDim.x) //j belong to this block, fetch data from shared memory, faster
			{
				nR=bs[j-base].R;
				nx=bs[j-base].x;
				ny=bs[j-base].y;
				nz=bs[j-base].z;
			}
			else //fetch from the global memory, slower
			{
				np=ptr_bubbles[j];
				nR=np->R;
				nx=np->x;
				ny=np->y;
				nz=np->z;
			}

			if(nz > up_limit) break; //exceed the detection range, quit
			if((sx-nx)*(sx-nx)+(sy-ny)*(sy-ny)+(sz-nz)*(sz-nz) < (sR+nR)*(sR+nR)) //they touched each other
			{
				if(!first_overlap)
				{
					sp->ip=j;
					atomicExch(&(ptr_bubbles[j]->iped),i);
					d_has_overlap=1;//atomicExch(&d_has_overlap,1);
					first_overlap=true;
				}
				else 
				{
					d_multi_tree=1;
					return; // finish detection for this bubble
				}	
			}
			--j;
		}
		//iterate to the right
		j=i+1;
		while(j<N_bubble)
		{
			if(j-base>=0 && j-base< blockDim.x) //j belong to this block
			{
				nR=bs[j-base].R;
				nx=bs[j-base].x;
				ny=bs[j-base].y;
				nz=bs[j-base].z;
			}
			else //fetch from the global memory
			{
				np=ptr_bubbles[j];
				nR=np->R;
				nx=np->x;
				ny=np->y;
				nz=np->z;
			}

			if(nz < low_limit) break; //exceed the detection range, quit
			if((sx-nx)*(sx-nx)+(sy-ny)*(sy-ny)+(sz-nz)*(sz-nz) < (sR+nR)*(sR+nR)) //they touched each other
			{	
				if(!first_overlap)
				{
					sp->ip=j;
					atomicExch(&(ptr_bubbles[j]->iped),i);
					d_has_overlap=1;//atomicExch(&d_has_overlap,1);
					first_overlap=true;
				}
				else 
				{
					d_multi_tree=1;
					return; // finish detection for this bubble
				}	
			}
			++j;
		}
	}
}

__global__ void multiTree2Chain(PBubble* ptr_bubbles, size_t N_bubble) //turn binary tree to chain
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x; //thread index
	if(i < N_bubble) //the thread index must be less than number of bubbles
	{
		size_t ipn=ptr_bubbles[i]->ip;
		if(ipn!=NULLD && ptr_bubbles[ipn]->iped!=i) 
		{
			ptr_bubbles[i]->ip=NULLD;	
			d_has_overlap=1; //which means we have to do the overlap detect again
		}
	}
}



__global__ void mergeBubbles(PBubble* ptr_bubbles, size_t N_bubble)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x; //thread index
	if(i < N_bubble) //the thread index must be less than number of bubbles
	{

		PBubble pb=ptr_bubbles[i];
		size_t ip=pb->ip;
		if(ip!=NULLD) //it collides with others
		{
			//determine it's in a chain or a loop
			//if it's in  a loop, we have to find out the biggest index in this loop
			//or it's in a chain, just check if this bubble is pointed by another bubble(determine the head node)
			int status=0;

			for (size_t ipn=ptr_bubbles[ip]->ip,max_ip=i>ip?i:ip;;ipn=ptr_bubbles[ipn]->ip)
			{	

				if(ipn==NULLD) //it's a chain
				{
					if(pb->iped==NULLD) status=1; //it's also the head of the chain
					break;
				}
				else if(ipn==i) //it's a loop
				{
					if(max_ip==i) status=2; //it can be also viewed as the beginning of the loop
					break;
				}
				max_ip=max_ip>ipn ? max_ip : ipn;
			}

			if(status) //keep on merging along the chain or loop
			{
				float V=pb->R3;	
				float Vt=0;
				float sumx=V*pb->x;
				float sumy=V*pb->y;
				float sumz=V*pb->z;
				PBubble pbn;
				for (size_t ipn=ip; ipn!=NULLD && ipn!=i; ipn=pbn->ip)
				{
					pbn=ptr_bubbles[ipn];
					Vt=pbn->R3;
					sumx += Vt*pbn->x;
					sumy += Vt*pbn->y;
					sumz += Vt*pbn->z;
					V += Vt;
					pbn->z=0;
				}
				pb->x = sumx/V;
				pb->y = sumy/V;
				pb->z = sumz/V;
				pb->R3=V;
				pb->R=powf(V,1.0/3.0);	
			}
		}
	}
}

__global__ void resetMergeMark(PBubble* ptr_bubbles, size_t N_bubble) //based on one-to-one collision
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x; //thread index
	if(i < N_bubble) //the thread index must be less than number of bubbles
	{
		ptr_bubbles[i]->ip=NULLD;
		ptr_bubbles[i]->iped=NULLD;
	}
}

/***************************************Functions in the CPU******************************************************/

double random(long* nextIn=NULL); //Random number generator
double random(long* nextIn) //Random number generator
{
	//static long In=(long)time(NULL);

	In=16807*(In%127773)-2836*(In/127773);
	if(In<0) In+=2147483647;
	if(nextIn) *nextIn=In;
	return (double)In/2147483647;
}

void loadConfig()
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0); // Choose which GPU to run on, change this on a multi-GPU system.
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		getchar();
		exit(1);
	}

	//initialize the default constant data in the GPU
	float h_Hm_max=1.0f;//horizontal move very step, unit mm
	float h_K_velocity=0.01f; //the coefficient in the formula u=(2*g*rho/9/eta)*R^2
	float h_K_escape=0.67f; //determine the critical condition for the bubble to escape


	//load configuration from data file
	FILE *fp=fopen("configure.txt","r");
	if(fp)
	{
		using std::map;
		using std::string;
		char name[50];//,value[50]; 
		float temp;
		int ch=0;
		map<string,float> mp;
		while (fscanf(fp,"%s %f",name,&temp)!=-1)
		{
			while((ch=getc(fp))!='\n' && ch!=-1){};
			mp[name]=temp;
		}
		//global constant in the CPU 
		N_max_bubble=(size_t)mp["N_max_bubble="];
		N_init=(size_t)mp["N_init="];
		N_max_step=(size_t)mp["N_max_step="];
		N_start_step=(size_t)mp["N_start_step="];
		N_steps_output=(int)mp["N_steps_output="];
		N_inject_rate=mp["N_inject_rate="];
		InitR=mp["InitR="];
		Width_container=mp["Width_container="];
		Height_container=mp["Height_container="];
		K_interval=mp["K_interval="];
		Init_height=mp["Init_height="];
		Block_size=(int)mp["Block_size="];
		Rand_seed=(int)mp["Rand_seed="];
		Init_var=(int)mp["Init_var="];
		Output_var=(int)mp["Output_var="];
		InitDistFolder=(int)mp["InitDistFolder="];
		//global constant in the GPU 
		h_Hm_max=mp["Hm_max="];
		h_K_velocity=mp["K_velocity="];
		h_K_escape=mp["K_escape="];
		
		fclose(fp);
	}
	else
	{
		printf("WARNING:\nThe configure.txt file doesn't exist in current directory!\nSo the program will use default values, which may be not good.\n\nPress any key to continue.\n\n");
		getchar();
	}
	//copy the constant to the GPU
	hipMemcpyToSymbol(HIP_SYMBOL(Hm_max),&h_Hm_max,sizeof(float));	
	hipMemcpyToSymbol(HIP_SYMBOL(K_velocity),&h_K_velocity,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(K_escape),&h_K_escape,sizeof(float));
	//copy the duplicated part
	hipMemcpyToSymbol(HIP_SYMBOL(Width_container_d),&Width_container,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Height_container_d),&Height_container,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(NULLD),&N_max_bubble,sizeof(size_t));

	//initialize the random number array in the GPU;
	if(h_Hm_max>0)
	{
		long* rand_array=(long*)malloc(N_max_bubble*sizeof(long));
		for(size_t i=0; i<N_max_bubble; ++i) random(rand_array+N_max_bubble-1-i);  
		long* d_rands;
		hipMalloc(&d_rands,N_max_bubble*sizeof(long));
		hipMemcpy(d_rands,rand_array,N_max_bubble*sizeof(long),hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(rands),&d_rands,sizeof(long *));
		free(rand_array);
	}
	if(Rand_seed) In=Rand_seed;
	else In=(long)time(NULL);
}

void initContainer(thrust::device_vector<PBubble>& vec_bubbles) //initialize the container with given distribution
{
	double X;
	Bubble b;
	Bubble *pb;
	/*load the distribution generator from dll file*/
	SetCurrentDirectory("InitDistribution");
	char searchScope[10];
	sprintf(searchScope,"%d.*",InitDistFolder);
	WIN32_FIND_DATA fd;
	HANDLE hFind = ::FindFirstFile(searchScope, &fd);
	bool found=false;
	if ( hFind != INVALID_HANDLE_VALUE )
	{
		do{
			if (fd.dwFileAttributes & FILE_ATTRIBUTE_DIRECTORY )//Ŀ¼
			{
				printf("The folder you designated is ");
				printf(fd.cFileName);
				printf("\n\n");
				SetCurrentDirectory(fd.cFileName);
				found=true;
				break;
			}
		}while (::FindNextFile(hFind, &fd));
		::FindClose(hFind);
	}
	if(!found)
	{
		printf("Cann't find the folder index %d you designated under 'InitDistribution'\n",InitDistFolder);
		printf("You must provide the right folder containing the right dll file to determine the initial distribution\n");
		getchar();
		exit(1);
	}

	HINSTANCE hdll=::LoadLibrary("Size.dll");
	if(!hdll)
	{
		printf("Error! the Size.dll is missing, which defines the initial distribution of the system.\n");
		getchar();
		exit(1);
	}
	typedef double (* DllFunc)(void); 
	DllFunc SizeofBubble = (DllFunc)GetProcAddress(hdll, "fnSize"); 
	if(!SizeofBubble)
	{
		printf("Error! Cannot get function 'fnSize' from Size.dll\n");
		getchar();
		exit(1);
	}
	/*end loading dll file*/
	
	for(size_t i=0;i<N_init;++i) 
	{
		X=SizeofBubble();
		//spread N_init bubbles in the container UNIFORMLY
		if(Init_var==0) // X is R
		{
			b.R=(float)X;
			b.R3=(float)(X*X*X);
		}
		else //X is V
		{
			b.R3=(float)(X/K_SPHERE);
			b.R=(float)pow(X/K_SPHERE,1.0/3.0);
		}

		b.x=(float)(b.R+random()*(Width_container-2*b.R));
		b.y=(float)(b.R+random()*(Width_container-2*b.R));
		b.z=(float)(b.R+random()*(Init_height-2*b.R));
		hipMalloc(&pb,sizeof(Bubble));
		vec_bubbles[N_bubble]=pb;
		hipMemcpy(pb,&b,sizeof(Bubble),hipMemcpyHostToDevice);		
		++N_bubble;
	}
	if(SizeofBubble) ::FreeLibrary(hdll); 
	//SetCurrentDirectory("../..");
}

void addNewBubbles(thrust::device_vector<PBubble>& vec_bubbles)
{
	//add some new bubbles from the bottom of the container
	int N_inject=(int)N_inject_rate;
	float p=N_inject_rate-N_inject;//the decimal part
	if(p && random()<p) ++N_inject; //use random number to determine whether to add one more particle
	//randomly spread these injected bubbles
	size_t max_ix=(size_t)((Width_container/InitR-4)/(2+K_interval)+1); //max index of the possible positions of the injected bubble
	Bubble b;
	Bubble *pb;
	for(int n=0; n < N_inject; ++n) //push N_inject bubbles into the container
	{		
		//generate random integer pairs between [1, max_ix]
		size_t ix=(size_t)(random()*(max_ix-0.1)+1); 
		size_t iy=(size_t)(random()*(max_ix-0.1)+1);  	
		b.R = InitR;
		b.R3=InitR*InitR*InitR;
		b.x = (3+(2+K_interval)*(ix-1))*InitR;
		b.y = (3+(2+K_interval)*(iy-1))*InitR;
		b.z = InitR;

		if(N_reusable)	--N_reusable; //try to reuse the memory in GPU, which will save time of memory allocation
		else
		{
			//allocate memory for this bubble
			hipMalloc(&pb,sizeof(Bubble));
			vec_bubbles[N_bubble]=pb;		
		}
		hipMemcpy(vec_bubbles[N_bubble],&b,sizeof(Bubble),hipMemcpyHostToDevice);		
		++N_bubble;
	}
}

void sortBubbles(thrust::device_vector<PBubble>& vec_bubbles) //sort the bubbles by their height, and retrieve memories
{
	thrust::sort(vec_bubbles.begin(),vec_bubbles.begin()+N_bubble,compare_height()); 
	hipStreamSynchronize(0);
	//try to retrieve some allocated data
	size_t index=N_bubble;
	PBubble pb=NULL;
	float z;
	do 
	{
		--index;
		if(index==0)
		{
			printf("\n\nWARNING:\nNo colliable bubble left! The simulation terminated earlier than expected!\n" \
				"Please adjust the parameters and run again.\n\nPress any key to continue...\n");
			getchar();
			exit(1);
		}
		pb=vec_bubbles[index];
		hipMemcpy(&z,&(pb->z),sizeof(float),hipMemcpyDeviceToHost);
	} while (z==0);
	++index;
	N_reusable+=N_bubble-index;
	N_bubble=index;
}

int overlapDetect(thrust::device_vector<PBubble>& vec_bubbles) //detect and mark collisions in pairs
{
	sortBubbles(vec_bubbles); //sort before we can detect overlap
	resetMergeMark<<<grid_size, Block_size>>>(VEC_D,N_bubble); //must reset the links first
	checkOverlap<<<grid_size, Block_size, Block_size*sizeof(Bubble)>>>(VEC_D,N_bubble);
	hipStreamSynchronize(0); //wait until all threads end
	int h_has_overlap=0,h_multi_tree=0;
	int reset=0;
	hipMemcpyFromSymbol(&h_has_overlap,HIP_SYMBOL(d_has_overlap),sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_has_overlap),&reset,sizeof(int)); //reset the indication in the GPU to zero

	if(h_has_overlap) //we need to find out what kind of overlap
	{
		multiTree2Chain<<<grid_size, Block_size>>>(VEC_D,N_bubble);
		hipStreamSynchronize(0); //wait for all threads end
		hipMemcpyFromSymbol(&h_has_overlap,HIP_SYMBOL(d_has_overlap),sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_has_overlap),&reset,sizeof(int)); //reset the indication in the GPU to zero

		hipMemcpyFromSymbol(&h_multi_tree,HIP_SYMBOL(d_multi_tree),sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(d_multi_tree),&reset,sizeof(int)); //reset the indication in the GPU to zero

		if(h_has_overlap||h_multi_tree) return 2; //there are multi trees that need repeat detection
		return 1; 
	}
	return 0; //there's no ovelap
}

void moveBubbles(thrust::device_vector<PBubble>& vec_bubbles, size_t& N_step)
{
	moveBubbles<<<grid_size, Block_size>>>(VEC_D,N_bubble); //move bubbles
	hipStreamSynchronize(0); //wait until all threads end
	int h_has_overlap=0;
	int reset=0;
	hipMemcpyFromSymbol(&h_has_overlap,HIP_SYMBOL(d_has_overlap),sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_has_overlap),&reset,sizeof(int)); //reset the indicator in the GPU to zero
	if(h_has_overlap) //here it means at least one bubble starts getting out of water
	{
		printf("\nCaution!\nAt least one bubble starts getting out of water at step=%d, please adjust the N_max_step or K_velocity, then restart the simulation.\n",N_step);
	}
}

int main()
{
	loadConfig(); //load some constants
	clock_t pre_time=clock(),now_time;
	thrust::device_vector<PBubble> vec_bubbles(N_max_bubble, NULL); //vector which contains pointers to the real data
	int overlap_status=0;
	initContainer(vec_bubbles);
	FILE* fp=fopen("status.txt","w"); //prepare to dump the runtime information
	if(fp==NULL)
	{
		printf("Cannot write file status.txt!");
		getchar();
		exit(1);
	}
	fprintf(fp,"width= %.4f\t\theight= %.4f\t\tnumber_snapshot= %d\t\tdata_type= %d\n",Width_container,Height_container,(N_max_step-N_start_step)/N_steps_output+1,Output_var);
	fprintf(fp,"*********************************************************************************************************\n\n");
	
	Bubble b;
	if(N_start_step==0) //we need to output the initial state
	{	
		fprintf(fp,"step= %d\t\tbubble_number= %d\n",0,N_bubble);
		for(size_t i=0; i<N_bubble; ++i)
		{
			hipMemcpy(&b,vec_bubbles[i],sizeof(Bubble),hipMemcpyDeviceToHost); //copy the data from the GPU to CPU
			if(Output_var==0) fprintf(fp,"% .4f\t\t\t% .4f\t\t\t% .4f\t\t\t% .4f\n", b.x, b.y, b.z, b.R);
			else fprintf(fp,"% .4f\t\t\t% .4f\t\t\t% .4f\t\t\t% .4f\n", b.x, b.y, b.z, K_SPHERE*b.R3);
		}			
		fprintf(fp,"*********************************************************************************************************\n\n\n");
		//tick the time
		now_time=clock();
		printf("the %dth step has finished, %d bubbles costing %f secs.\n",0,N_bubble,(now_time-pre_time)/(float)CLOCKS_PER_SEC);
		pre_time=now_time;
	}
	for (size_t step=1; step <= N_max_step; ++step) //repeat the simulation until N_max_step
	{
		if(N_inject_rate>0) addNewBubbles(vec_bubbles); //pump in some new bubbles
		moveBubbles(vec_bubbles,step);
		do  //the Overlap Detection depends on the ordered sequence
		{
			overlap_status=overlapDetect(vec_bubbles);
			if (overlap_status)
			{
				//deal with the merge
				mergeBubbles<<<grid_size, Block_size>>>(VEC_D,N_bubble); //merge bubbles, and have eliminated some bubbles
				hipStreamSynchronize(0); //wait until all threads end
			}	
		}while (overlap_status==2);  

		//output information
		if((step-N_start_step)%N_steps_output==0 && step>=N_start_step)
		{
			//dump the status of whole container			
			fprintf(fp,"step= %d\t\tbubble_number= %d\n",step,N_bubble);
			for(size_t i=0; i<N_bubble; ++i)
			{
				hipMemcpy(&b,vec_bubbles[i],sizeof(Bubble),hipMemcpyDeviceToHost); //copy the data from the GPU
				if(Output_var==0) fprintf(fp,"% .4f\t\t\t% .4f\t\t\t% .4f\t\t\t% .4f\n", b.x, b.y, b.z, b.R);
				else fprintf(fp,"% .4f\t\t\t% .4f\t\t\t% .4f\t\t\t% .4f\n", b.x, b.y, b.z, K_SPHERE*b.R3);
			}			
			fprintf(fp,"*********************************************************************************************************\n\n\n");
			//tick the time
			now_time=clock();
			printf("the %dth step has finished, %d bubbles costing %f secs.\n",step,N_bubble,(now_time-pre_time)/(float)CLOCKS_PER_SEC);
			pre_time=now_time;
		}
	}

	fclose(fp);
	printf("\nTotal Elapsed Time:%f secs.\n",clock()/(float)CLOCKS_PER_SEC);
	printf("Press any key to continue!\n");
	getchar();
	//SetCurrentDirectory("../..");
	ShellExecute(NULL,"open","VisualBubbles.exe",NULL,NULL,SW_SHOWNORMAL); //this is windows platform specific
	return 0;
}